#include "hip/hip_runtime.h"
#include "img_proc.hpp"

// CUDA 커널 함수
__global__ void color_to_hsv(const unsigned char* bgr, unsigned char* hsv, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = (y * width + x) * 3;
        float b = bgr[index + 0] / 255.0f;
        float g = bgr[index + 1] / 255.0f;
        float r = bgr[index + 2] / 255.0f;

        float max_val = fmaxf(fmaxf(r, g), b);
        float min_val = fminf(fminf(r, g), b);
        float delta = max_val - min_val;

        float h = 0.0f, s = 0.0f, v = max_val;

        if (delta > 0.00001f) {
            s = delta / max_val;

            if (r >= max_val)
                h = (g - b) / delta;
            else if (g >= max_val)
                h = 2.0f + (b - r) / delta;
            else
                h = 4.0f + (r - g) / delta;

            h *= 60.0f;
            if (h < 0.0f) h += 360.0f;
        }

        int hsv_index = (y * width + x) * 3;
        hsv[hsv_index + 0] = static_cast<unsigned char>(h / 2);       // H value (0-180)
        hsv[hsv_index + 1] = static_cast<unsigned char>(s * 255);     // S value (0-255)
        hsv[hsv_index + 2] = static_cast<unsigned char>(v * 255);     // V value (0-255)
    }
}

__global__ void filter_red_pixels(const unsigned char* hsv, unsigned char* red_mask, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = (y * width + x) * 3;
        unsigned char h = hsv[index + 0];
        unsigned char s = hsv[index + 1];
        unsigned char v = hsv[index + 2];

        if (((h <= 10 || h >= 160) && s >= 100 && v >= 100)) {
            red_mask[y * width + x] = 255;
        } else {
            red_mask[y * width + x] = 0;
        }
    }
}

// Constructor
Img_proc::Img_proc()
    : SPIN_RATE(100),
      img_proc_line_det_(false),
      gradient_(0)
{
    // RealSense 파이프라인 초기화
    cfg.enable_stream(RS2_STREAM_COLOR, realsense_width, realsense_height, RS2_FORMAT_BGR8, realsense_color_fps);
    cfg.enable_stream(RS2_STREAM_DEPTH, realsense_width, realsense_height, RS2_FORMAT_Z16, realsense_depth_fps);

    hipStreamCreate(&stream);
}

// 소멸자
Img_proc::~Img_proc()
{
    // CUDA 스트림 파괴
    hipStreamDestroy(stream);
}

void Img_proc::on_trackbar(int, void *)
{
    // Function body if required.
}

void Img_proc::create_color_range_trackbar(const std::string &window_name)
{
    cv::createTrackbar("Hue Lower", window_name, &lowerH, 179, on_trackbar);
    cv::createTrackbar("Hue Upper", window_name, &upperH, 179, on_trackbar);
    cv::createTrackbar("Saturation Lower", window_name, &lowerS, 255, on_trackbar);
    cv::createTrackbar("Saturation Upper", window_name, &upperS, 255, on_trackbar);
    cv::createTrackbar("Value Lower", window_name, &lowerV, 255, on_trackbar);
    cv::createTrackbar("Value Upper", window_name, &upperV, 255, on_trackbar);
}

std::tuple<cv::Mat, cv::Mat> Img_proc::ROI_Line(const cv::Mat &input_frame, const cv::Mat &ori_frame) // 관심영역 설정하는 코드 -- 육상
{
    cv::Mat draw_frame = ori_frame.clone();
    cv::Mat mask = cv::Mat::zeros(input_frame.size(), CV_8UC1);

    cv::rectangle(mask, cv::Point(250, 480), cv::Point(698, 0), cv::Scalar(255), -1);

    cv::Mat circleMask = cv::Mat::zeros(input_frame.size(), CV_8UC1);

    cv::circle(circleMask, cv::Point(490, 500), 500, cv::Scalar(255), -1);
    cv::bitwise_and(mask, circleMask, mask);

    int circle_center_x = 490;
    int circle_center_y = 600;
    int radius = 500;

    for (int x = 250; x <= 698; x++)
    {

        double y_positive = circle_center_y + std::sqrt(radius * radius - (x - circle_center_x) * (x - circle_center_x));
        double y_negative = circle_center_y - std::sqrt(radius * radius - (x - circle_center_x) * (x - circle_center_x));

    }

    cv::Mat roi;
    ori_frame.copyTo(roi, mask);

    return {roi, draw_frame};
}

cv::Mat Img_proc::ROI_Rectangle(const cv::Mat &input_frame, int y_start, int y_end, int x_start, int x_end)
{
    cv::Mat result = input_frame.clone();
    
    // 좌측 영역을 하얗게 채움
    cv::rectangle(result, cv::Point(0, 0), cv::Point(x_start, result.rows), cv::Scalar(255, 255, 255), -1);
    
    // 우측 영역을 하얗게 채움
    cv::rectangle(result, cv::Point(x_end, 0), cv::Point(result.cols, result.rows), cv::Scalar(255, 255, 255), -1);
    
    // ROI 영역의 경계를 빨간색으로 표시 (선택사항)
    cv::rectangle(result, cv::Point(x_start, y_start), cv::Point(x_end, y_end), cv::Scalar(0, 0, 255), 2);

    return result;
}

std::tuple<cv::Mat, cv::Mat> Img_proc::extract_color(cv::Mat &input_frame, const cv::Scalar &lower_bound, const cv::Scalar &upper_bound)
{
    cv::Mat hsv;
    cv::cvtColor(input_frame, hsv, cv::COLOR_BGR2HSV);

    cv::Mat mask;
    cv::inRange(hsv, lower_bound, upper_bound, mask);

    cv::Mat color_extracted;
    cv::bitwise_and(input_frame, input_frame, color_extracted, mask);

    return {color_extracted, input_frame}; 
}

std::tuple<cv::Mat, float, int, double>
Img_proc::detect_Line_areas(const cv::Mat &input_frame, const cv::Mat &origin_frame, const cv::Scalar &contour_color, int threshold_value)
{
    // Clone the origin frame to draw results
    cv::Mat ori_frame = origin_frame.clone();

    // Upload the input frame to GPU
    cv::cuda::GpuMat d_frame(input_frame);

    // Convert to grayscale on GPU
    cv::cuda::GpuMat d_gray;
    cv::cuda::cvtColor(d_frame, d_gray, cv::COLOR_BGR2GRAY);

    // Apply threshold on GPU
    cv::cuda::GpuMat d_binary;
    cv::cuda::threshold(d_gray, d_binary, threshold_value, max_value, cv::THRESH_BINARY);  // 추가된 인자


    // Download the binary image from GPU to CPU for further processing
    cv::Mat binary;
    d_binary.download(binary);

    // Find contours on the CPU (currently OpenCV does not support contour finding on GPU)
    std::vector<std::vector<cv::Point>> contours;
    cv::findContours(binary, contours, cv::RETR_EXTERNAL, cv::CHAIN_APPROX_SIMPLE);

    std::vector<cv::Point> top_contour;

    bool foundLargeContour = false;
    double topmost_y = std::numeric_limits<double>::max();
    bool has_white_now = false;

    float SM_angle = 0;
    float Rnd_angle = 0;
    float Line_Angle = 0;
    int line_area = 0;

    cv::Point top_center, bottom_center, left_center, right_center;

    bool &has_prev = has_white_prev;
    cv::Point &center_now = center_now_white;

    for (const auto &contour : contours)
    {
        line_area = cv::contourArea(contour);

        if (line_area < 2500 && line_area > LINE_AREA)
        {
            cv::Moments m = cv::moments(contour);
            foundLargeContour = true;
            line_condition_count = 0;
            if (m.m00 == 0)
                continue;

            cv::Point center(m.m10 / m.m00, m.m01 / m.m00);
            if (center.y < topmost_y)
            {
                topmost_y = center.y;
                top_contour = contour;
                center_now = center;
            }
            has_white_now = true;
        }
        else if (line_area < LINE_AREA)
        {
            line_condition_count++;
            if (line_condition_count >= 15)
            {
                foundLargeContour = false;
                has_white_now = false;
            }
        }
    }

    if (!top_contour.empty())
    {
        top_contour_area = cv::contourArea(top_contour);
        cv::line(ori_frame, center_now, cv::Point(490, 480), contour_color, 2);

        float deltaY = center_now.y - 480;   
        float deltaX = center_now.x - 490;    

        float radians = atan2(deltaY, deltaX);

        float adjustedAngle = radians * (180.0 / CV_PI);

        Rnd_angle = -90 - adjustedAngle;
    }

    if (has_prev && !has_white_now && center_now.x < 490)
    {
        int8_t tmp_delta_x = 1;
        delta_x_ = tmp_delta_x;
        tmp_delta_x = 0;
        std::cout << "Line area disappeared to the left\n";
    }
    else if (has_prev && !has_white_now && center_now.x > 490)
    {
        int8_t tmp_delta_x = -1;
        delta_x_ = tmp_delta_x;
        tmp_delta_x = 0;
        std::cout << "Line area disappeared to the right\n";
    }

    has_prev = has_white_now;

    if (!top_contour.empty())
    {
        std::vector<cv::Point> approx;
        double epsilon = 0.02 * cv::arcLength(top_contour, true);
        cv::approxPolyDP(top_contour, approx, epsilon, true);

        int numVertices = approx.size(); // 근사화된 컨투어의 꼭지점 수를 얻음

        cv::RotatedRect min_area_rect = cv::minAreaRect(top_contour);

        float width = min_area_rect.size.width;
        float height = min_area_rect.size.height;

        float long_len = (width > height) ? width : height;
        float short_len = (width > height) ? height : width;

        cv::Point2f vertices[4];
        min_area_rect.points(vertices);


                    for (int i = 0; i < 4; ++i)
            cv::line(ori_frame, vertices[i], vertices[(i + 1) % 4], contour_color, 3);

        // Line angle
        if (  short_len * 1.2 < long_len && short_len * 5 > long_len && numVertices > 3 && numVertices < 5)
        {
            if (min_area_rect.size.width < min_area_rect.size.height)
            {
                SM_angle = -min_area_rect.angle;
            }
            else
            {
                SM_angle = -min_area_rect.angle + 90;
            }



            // cv::imshow("white binary", binary);
            // cv::moveWindow("white binary", 700, 540);
        }

        if (Rnd_angle > 0)
        {
            Line_Angle = (SM_angle + Rnd_angle) * 0.5;
        }
        else if (Rnd_angle <= 0)
        {
            Line_Angle = (SM_angle + (3 * Rnd_angle));
        }

        //--------------------------------------------------------------------- frame interface -------------------------------------------------------------------------

        if (has_white_now)
        {
            cv::putText(ori_frame, "MODE : " + Str_LINE_MODE, cv::Point(490 + 50, 25), cv::FONT_HERSHEY_SIMPLEX, 0.7, contour_color, 2);

        }
        if (!has_white_now)
        {
            cv::putText(ori_frame, "MODE : " + Str_NO_LINE_MODE, cv::Point(490 + 50, 25), cv::FONT_HERSHEY_SIMPLEX, 0.7, contour_color, 2);
        }

        cv::putText(ori_frame, "Rnd_Angle : " + std::to_string(Rnd_angle), cv::Point(10, 50), cv::FONT_HERSHEY_SIMPLEX, 0.6, contour_color, 2);
        cv::putText(ori_frame, "SM_angle : " + std::to_string(SM_angle), cv::Point(10, 25), cv::FONT_HERSHEY_SIMPLEX, 0.6, contour_color, 2);
        cv::putText(ori_frame, "Line angle : " + std::to_string(SM_angle + Rnd_angle), cv::Point(10, 75), cv::FONT_HERSHEY_SIMPLEX, 0.6, contour_color, 2);
        cv::putText(ori_frame, "Vertice : " + std::to_string(numVertices), cv::Point(10, 300), cv::FONT_HERSHEY_SIMPLEX, 0.6, contour_color, 2);
        cv::putText(ori_frame, "Area : " + std::to_string(top_contour_area), cv::Point(10, 455), cv::FONT_HERSHEY_SIMPLEX, 0.7, contour_color, 2);

        topmost_point = *std::min_element(top_contour.begin(), top_contour.end(),
                                          [](const cv::Point &a, const cv::Point &b)
                                          {
                                              return a.y < b.y;
                                          });

        bottommost_point = *std::max_element(top_contour.begin(), top_contour.end(),
                                             [](const cv::Point &a, const cv::Point &b)
                                             {
                                                 return a.y < b.y;
                                             });
    }

    return std::make_tuple(ori_frame, Line_Angle, line_area, delta_x_);
}

std::tuple<cv::Mat, int, cv::Point, cv::Point, float, cv::Point, std::vector<cv::Point>, int>
Img_proc::detect_Huddle_areas(const cv::Mat &input_frame, const cv::Mat &origin_frame, const cv::Scalar &contour_color, int threshold_value)
{
    // Clone the origin frame to draw results
    cv::Mat ori_frame = origin_frame.clone();

    // Upload the input frame to GPU
    cv::cuda::GpuMat d_frame(input_frame);

    // Convert to grayscale on GPU
    cv::cuda::GpuMat d_gray;
    cv::cuda::cvtColor(d_frame, d_gray, cv::COLOR_BGR2GRAY);

    // Apply threshold on GPU
    cv::cuda::GpuMat d_binary;
    cv::cuda::threshold(d_gray, d_binary, threshold_value, max_value, cv::THRESH_BINARY);  // 추가된 인자


    // Download the binary image from GPU to CPU for further processing
    cv::Mat binary;
    d_binary.download(binary);

    // Find contours on the CPU (currently OpenCV does not support contour finding on GPU)
    std::vector<std::vector<cv::Point>> contours;
    cv::findContours(binary, contours, cv::RETR_EXTERNAL, cv::CHAIN_APPROX_SIMPLE);

    std::vector<cv::Point> top_contour;

    bool foundLargeContour = false;
    double topmost_y = std::numeric_limits<double>::max();
    double distance_huddle = 0;
    bool has_yellow_now = false;

    float SM_angle = 0;
    float Rnd_angle = 0;
    float huddle_angle = 0;
    float Line_Angle = 0;
    float corner_angle = 0;
    int huddle_area = 0;

    cv::Point huddle_center = cv::Point(0, 0);

    cv::Point top_center, bottom_center, left_center, right_center;
    cv::Point &center_now = center_now_yellow;

    for (const auto &contour : contours)
    {
        /// Huddle
        huddle_area = cv::contourArea(contour);

        if (huddle_area > HUDDLE_AREA)
        {
            cv::Moments m = cv::moments(contour);
            foundLargeContour = true;
            line_condition_count = 0;
            if (m.m00 == 0)
                continue;

            cv::Point center(m.m10 / m.m00, m.m01 / m.m00);
            if (center.y < topmost_y)
            {
                topmost_y = center.y;
                top_contour = contour;
                center_now = center;
                distance_huddle = 480 - topmost_y;
            }
            has_yellow_now = true;
        }
    }

    if (!top_contour.empty())
    {
        top_contour_area = cv::contourArea(top_contour);
        cv::line(ori_frame, center_now, cv::Point(424, 480), contour_color, 2);

        float deltaY = center_now.y - 480;
        float deltaX = center_now.x - 424;

        float radians = atan2(deltaY, deltaX);

        float adjustedAngle = radians * (180.0 / CV_PI);

        Rnd_angle = -90 - adjustedAngle;
    }

    if (!top_contour.empty())
    {
        std::vector<cv::Point> approx;
        double epsilon = 0.02 * cv::arcLength(top_contour, true);
        cv::approxPolyDP(top_contour, approx, epsilon, true);

        int numVertices = approx.size();

        cv::RotatedRect min_area_rect = cv::minAreaRect(top_contour);

        float width = min_area_rect.size.width;
        float height = min_area_rect.size.height;

        float long_len = (width > height) ? width : height;
        float short_len = (width > height) ? height : width;

        cv::Point2f vertices[4];
        min_area_rect.points(vertices);

        for (int i = 0; i < 4; ++i)
            cv::line(ori_frame, vertices[i], vertices[(i + 1) % 4], contour_color, 3);


        if (short_len * 1.5 < long_len)
        {
            if (min_area_rect.size.width < min_area_rect.size.height)
            {
                huddle_angle = -min_area_rect.angle + 90;
            }
            else
            {
                huddle_angle = -min_area_rect.angle;
            }
            huddle_center = min_area_rect.center;
            cv::circle(ori_frame, huddle_center, 2, contour_color, -1, 8);
        }

        if (Rnd_angle > 0)
        {
            Line_Angle = Rnd_angle * 0.5;
        }
        else if (Rnd_angle <= 0)
        {
            Line_Angle = (3 * Rnd_angle);
        }

        //--------------------------------------------------------------------- frame interface -------------------------------------------------------------------------

        cv::putText(ori_frame, "Rnd_Angle : " + std::to_string(Rnd_angle), cv::Point(10, 50), cv::FONT_HERSHEY_SIMPLEX, 0.6, contour_color, 2);
        cv::putText(ori_frame, "Hurdle Angle : " + std::to_string(huddle_angle), cv::Point(10, 25), cv::FONT_HERSHEY_SIMPLEX, 0.6, contour_color, 2);
        cv::putText(ori_frame, "Line angle : " + std::to_string(Line_Angle), cv::Point(10, 75), cv::FONT_HERSHEY_SIMPLEX, 0.6, contour_color, 2);
        cv::putText(ori_frame, "Vertice : " + std::to_string(numVertices), cv::Point(10, 300), cv::FONT_HERSHEY_SIMPLEX, 0.6, contour_color, 2);
        cv::putText(ori_frame, "Area : " + std::to_string(top_contour_area), cv::Point(10, 455), cv::FONT_HERSHEY_SIMPLEX, 0.7, contour_color, 2);

        topmost_point = *std::min_element(top_contour.begin(), top_contour.end(),
                                          [](const cv::Point &a, const cv::Point &b)
                                          {
                                              return a.y < b.y;
                                          });

        bottommost_point = *std::max_element(top_contour.begin(), top_contour.end(),
                                             [](const cv::Point &a, const cv::Point &b)
                                             {
                                                 return a.y < b.y;
                                             });
    }

    return std::make_tuple(ori_frame, Line_Angle, topmost_point, bottommost_point, huddle_angle, huddle_center, top_contour, huddle_area);
}

cv::Point Img_proc::detect_green_point(const cv::Mat &input_frame, int threshold_value)
{
    // Upload the input frame to GPU
    cv::cuda::GpuMat d_frame(input_frame);

    // Convert to grayscale on GPU
    cv::cuda::GpuMat d_gray;
    cv::cuda::cvtColor(d_frame, d_gray, cv::COLOR_BGR2GRAY);

    // Apply threshold on GPU
    cv::cuda::GpuMat d_binary;
    cv::cuda::threshold(d_gray, d_binary, threshold_value, max_value, cv::THRESH_BINARY);  // 추가된 인자


    // Download the binary image from GPU to CPU for further processing
    cv::Mat binary;
    d_binary.download(binary);

    // Find contours on the CPU (currently OpenCV does not support contour finding on GPU)
    std::vector<std::vector<cv::Point>> contours;
    cv::findContours(binary, contours, cv::RETR_EXTERNAL, cv::CHAIN_APPROX_SIMPLE);

    // Initialize variables
    std::vector<cv::Point> top_contour;
    double topmost_y = std::numeric_limits<double>::max();

    cv::Point topmost_point;

    for (const auto &contour : contours)
    {
            cv::Moments m = cv::moments(contour);

            if (m.m00 == 0)
                continue;

            cv::Point center(m.m10 / m.m00, m.m01 / m.m00);

            if (center.y < topmost_y)
            {
                topmost_y = center.y;
                top_contour = contour;
            }        
    }

    if (!top_contour.empty())
    {
        topmost_point = *std::min_element(top_contour.begin(), top_contour.end(),
                                          [](const cv::Point &a, const cv::Point &b) { return a.y < b.y; });
    }

    return topmost_point;
}

std::tuple<cv::Mat, cv::Point3f> Img_proc::Ball_Detect(cv::Mat color, cv::Mat depth_dist, int threshold_value) {
    // 깊이 값이 1000mm 이하인 영역만 마스크로 적용
    cv::Mat mask = depth_dist < 1000;

    // 원본 color 이미지를 mask를 사용하여 필터링
    cv::Mat output = color.clone();  // color.copyTo(output, mask);를 대체
    output.setTo(cv::Scalar(255, 255, 255), ~mask);

    // 화면 중앙 좌표 계산
    int center_x = output.cols / 2;
    int center_y = output.rows / 2;

    cv::Point3f ball_center(0, 0, 0); // 결과값으로 반환될 공의 좌표

    // HSV 색상 공간 변환
    cv::Mat hsv;
    cv::cvtColor(output, hsv, cv::COLOR_BGR2HSV);

    // 농구공 색상 범위 정의 (주황색)
    const cv::Scalar lower_orange(8, 60, 0);
    const cv::Scalar upper_orange(60, 255, 255);
    // 주황색 마스크 생성
    cv::Mat color_mask;
    cv::inRange(hsv, lower_orange, upper_orange, color_mask);

    // 노이즈 제거
    cv::Mat kernel = cv::getStructuringElement(cv::MORPH_ELLIPSE, cv::Size(5, 5));
    cv::morphologyEx(color_mask, color_mask, cv::MORPH_OPEN, kernel);
    cv::morphologyEx(color_mask, color_mask, cv::MORPH_CLOSE, kernel);

    // 윤곽선 검출
    std::vector<std::vector<cv::Point>> contours;
    cv::findContours(color_mask, contours, cv::RETR_EXTERNAL, cv::CHAIN_APPROX_SIMPLE);

    // 최소 원형도 및 면적 조건
    const double min_circularity = 0.7;
    const double min_area = 1000;
    const double max_area = 50000;

    double max_valid_area = 0;
    int max_valid_index = -1;

    for (int i = 0; i < contours.size(); i++) {
        double area = cv::contourArea(contours[i]);
        if (area < min_area || area > max_area) continue;

        cv::Point2f center;
        float radius;
        cv::minEnclosingCircle(contours[i], center, radius);

        // 원형도 계산
        double perimeter = cv::arcLength(contours[i], true);
        double circularity = 4 * CV_PI * area / (perimeter * perimeter);

        if (circularity > min_circularity && area > max_valid_area) {
            max_valid_area = area;
            max_valid_index = i;
        }
    }

    if (max_valid_index >= 0) {
        cv::Point2f center;
        float radius;
        cv::minEnclosingCircle(contours[max_valid_index], center, radius);

        // 원형 객체 그리기
        cv::circle(output, cv::Point(center.x, center.y), radius, cv::Scalar(0, 255, 0), 2);

        // 좌표 범위 확인 후 깊이 값 가져오기
        if (center.x >= 0 && center.x < depth_dist.cols && center.y >= 0 && center.y < depth_dist.rows) {
            float cz = depth_dist.at<uint16_t>(cv::Point(center.x, center.y)) * 0.001f;

            // 화면 중앙 기준 좌표 변환
            float cx = center.x - center_x;
            float cy = center.y - center_y;

            ball_center = cv::Point3f(cx, cy, cz);

            // 검출된 공의 중심 표시
            cv::circle(output, cv::Point(center.x, center.y), 5, cv::Scalar(0, 0, 255), -1);

            // 객체 정보 표시
            std::string info = "Ball: (" + std::to_string(int(ball_center.x)) + ", "
                + std::to_string(int(ball_center.y)) + ", "
                + std::to_string(ball_center.z) + "m)";
            cv::putText(output, info, cv::Point(10, 30), cv::FONT_HERSHEY_SIMPLEX, 0.7, cv::Scalar(0, 255, 0), 2);
        }
    }

    // ROI 영역 표시 (선택사항)
    cv::Rect roi(0, 0, output.cols, output.rows);
    cv::rectangle(output, roi, cv::Scalar(0, 255, 0), 2);

    return std::make_tuple(output, ball_center);
}

std::tuple<cv::Mat, cv::Point3f> Img_proc::Hoop_Detect(cv::Mat color, cv::Mat depth_dist, int threshold_value)
{

    int width = color.cols;
    int height = color.rows;

    unsigned char *d_color, *d_hsv, *d_red_mask;

    size_t color_size = width * height * 3 * sizeof(unsigned char);
    size_t mask_size = width * height * sizeof(unsigned char);

    // GPU 메모리 할당
    hipMalloc(&d_color, color_size);
    hipMalloc(&d_hsv, color_size);
    hipMalloc(&d_red_mask, mask_size);

    // 호스트에서 디바이스로 데이터 복사
    hipMemcpyAsync(d_color, color.data, color_size, hipMemcpyHostToDevice, stream);

    // CUDA 커널 실행 설정
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    // CUDA 커널 호출
    color_to_hsv<<<gridSize, blockSize, 0, stream>>>(d_color, d_hsv, width, height);
    
    // 필수적인 동기화 추가
    hipStreamSynchronize(stream);

    filter_red_pixels<<<gridSize, blockSize, 0, stream>>>(d_hsv, d_red_mask, width, height);

    // 또다시 동기화 추가
    hipStreamSynchronize(stream);

    // 결과를 호스트로 복사
    std::vector<unsigned char> red_mask_host(mask_size);
    hipMemcpyAsync(red_mask_host.data(), d_red_mask, mask_size, hipMemcpyDeviceToHost, stream);

    // CUDA 스트림 동기화
    hipStreamSynchronize(stream);

    // OpenCV Mat으로 변환
    cv::Mat red_mask_mat(height, width, CV_8UC1, red_mask_host.data());

    // 윤곽선 검출
    std::vector<std::vector<cv::Point>> contours;
    cv::findContours(red_mask_mat, contours, cv::RETR_EXTERNAL, cv::CHAIN_APPROX_SIMPLE);

    // 변수 초기화
    double max_area = 0;
    cv::RotatedRect largest_rect;
    bool detection_made = false;
    cv::Point3f hoop_center(0, 0, 0);

    for (const auto& contour : contours) {
        cv::RotatedRect min_area_rect = cv::minAreaRect(contour);
        float width_rect = min_area_rect.size.width;
        float height_rect = min_area_rect.size.height;

        double area = width_rect * height_rect;
        if (area > 5000 && 45000 > area) {
            float long_len = std::max(width_rect, height_rect);
            float short_len = std::min(width_rect, height_rect);
            float aspect_ratio = long_len / short_len;

            if (aspect_ratio >= 1.0 && aspect_ratio <= 2.5) {
                cv::Rect bbox = min_area_rect.boundingRect();
                bbox &= cv::Rect(0, 0, color.cols, color.rows);

                cv::Mat cropped = color(bbox);
                cv::Mat hsv_cropped;
                cv::cvtColor(cropped, hsv_cropped, cv::COLOR_BGR2HSV);

                cv::Mat mask1, mask2, red_mask;
                cv::inRange(hsv_cropped, cv::Scalar(159, 145, 0), cv::Scalar(180, 255, 255), mask1);
                //cv::inRange(hsv_cropped, cv::Scalar(160, 100, 100), cv::Scalar(180, 255, 255), mask2);
                red_mask = mask1;

                double red_pixels = cv::countNonZero(red_mask);
                double total_pixels = bbox.area();
                double red_ratio = red_pixels / total_pixels;

                if (0.1 < red_ratio && red_ratio < 0.4) {
                    if (area > max_area) {
                        max_area = area;
                        largest_rect = min_area_rect;
                        detection_made = true;
                    }
                }
            }
        }
    }

    cv::Mat output = color.clone();

    if (detection_made) {
        cv::Point2f vertices[4];
        largest_rect.points(vertices);
        for (int i = 0; i < 4; ++i)
            cv::line(output, vertices[i], vertices[(i + 1) % 4], cv::Scalar(0, 255, 0), 2);

        cv::Point2f center = largest_rect.center;

        // 깊이 값 추출
        float depth_value = depth_dist.at<uint16_t>(cv::Point(center.x, center.y)) * 0.001f;

        // 화면 중앙 기준 좌표 변환
        int center_x = output.cols / 2;
        int center_y = output.rows / 2;

        float cx = center.x - center_x;
        float cy = center_y - center.y;

        hoop_center = cv::Point3f(cx, cy, depth_value);

        // 중심점 표시
        cv::circle(output, center, 5, cv::Scalar(0, 0, 255), -1);

        // 정보 출력
        std::string info = "Hoop: (" + std::to_string(int(hoop_center.x)) + ", "
            + std::to_string(int(hoop_center.y)) + ", "
            + std::to_string(hoop_center.z) + "m)";
        cv::putText(output, info, cv::Point(10, 30), cv::FONT_HERSHEY_SIMPLEX, 0.7, cv::Scalar(0, 255, 0), 2);
    }

    // 메모리 해제
    hipFree(d_color);
    hipFree(d_hsv);
    hipFree(d_red_mask);
    // hipStreamDestroy(stream);

    return std::make_tuple(output, hoop_center);
}

void Img_proc::realsense_thread() {

    try {
        pipe.start(cfg);
    } catch (const rs2::error& e) {
        std::cerr << "Failed to open the RealSense camera: " << e.what() << std::endl;
        return;
    }

    const auto window_name = "RealSense Ball Frame";
    const auto window_name_hoop = "RealSense Hoop Frame";
    cv::namedWindow(window_name, cv::WINDOW_AUTOSIZE);
    cv::namedWindow(window_name_hoop, cv::WINDOW_AUTOSIZE);
    const auto window_name_hsv = "HSV Binary Image";
    cv::namedWindow(window_name_hsv, cv::WINDOW_AUTOSIZE);
    create_color_range_trackbar(window_name_hsv);

    rs2::align align_to(RS2_STREAM_COLOR);

    int White_count = 0;
    int Yellow_count = 0;
    int noline_count = 0;
    int ball_detected_count = 0;
    int hoop_detected_count = 0;
    const int detection_threshold = 3;

    cv::Mat colorMat, depthMat, Line_frame;

    int frameCounter = 0;
    // img_proc_mode = Interest_Object::Ball;

    while (ros::ok() && cv::waitKey(1) < 0) {
        frameCounter++;

        // Retrieve and align frames
        rs2::frameset data = pipe.wait_for_frames();
        data = align_to.process(data);

        rs2::frame color_frame = data.get_color_frame();
        rs2::depth_frame depth_frame = data.get_depth_frame();

        if (!color_frame || !depth_frame) {
            continue;
        }

        // Create color and depth matrices
        colorMat = cv::Mat(cv::Size(realsense_width, realsense_height), CV_8UC3, (void*)color_frame.get_data(), cv::Mat::AUTO_STEP);
        depthMat = cv::Mat(cv::Size(realsense_width, realsense_height), CV_16UC1, (void*)depth_frame.get_data(), cv::Mat::AUTO_STEP);

        // cv::Rect roi(100, 0, realsense_width - 100, realsense_height);
        // roi_line_frame = colorMat(roi);
        // roi_depth_mat = depthMat(roi);

        bool object_detected = false;
        // Update detection mode based on previous detections
        if (this->Get_img_proc_ball_done_()) {
            img_proc_mode = Interest_Object::Hoop;
            this->Set_img_proc_ball_done_(false);
            ball_detected_count = 0;
        } else if (this->Get_img_proc_hoop_done_()) {
            img_proc_mode = Interest_Object::Huddle;
            this->Set_img_proc_hoop_done_(false);
            hoop_detected_count = 0;
        } else if (this->Get_img_proc_huddle_done_()) {
            img_proc_mode = Interest_Object::Ball;
            this->Set_img_proc_huddle_done_(false);
        }

        // Object detection based on mode
        switch (img_proc_mode) {
            case Interest_Object::Ball:
                if (frameCounter % 3 == 0 || Get_UD_NeckAngle_() == PICK_NECK) {
                    auto Ball = Ball_Detect(colorMat, depthMat, threshold_value_black);
                    cv::Point3f Ball_center = std::get<1>(Ball);

                    if ((Ball_center.x != 0 || Ball_center.y != 0 || Ball_center.z != 0)) {
                        ball_detected_count++;
                        if (ball_detected_count >= detection_threshold) {
                            this->Set_img_proc_ball_det_(true);
                            this->Set_ball_x(Ball_center.x);
                            this->Set_ball_y(Ball_center.y);
                            this->Set_ball_z(Ball_center.z);
                            object_detected = true;

                            
                            cv::imshow(window_name, std::get<0>(Ball));
                            cv::moveWindow(window_name, 900, 700);
                        }
                    } else {
                        ball_detected_count = 0;
                        this->Set_img_proc_ball_det_(false);
                    }
                }
                break;

            case Interest_Object::Hoop: 
                if (Get_UD_NeckAngle_() == HOOP_NECK) {
                // if (0) {
                    auto Hoop = Hoop_Detect(colorMat, depthMat, threshold_value_black);
                    cv::Point3f Hoop_center = std::get<1>(Hoop);

                    if ((Hoop_center.x != 0 || Hoop_center.y != 0 || Hoop_center.z != 0)) {
                        hoop_detected_count++;
                        if (hoop_detected_count >= detection_threshold) {
                            this->Set_img_proc_hoop_det_(true);
                            this->Set_hoop_x(Hoop_center.x);
                            this->Set_hoop_y(Hoop_center.y);
                            this->Set_hoop_z(Hoop_center.z);
                            object_detected = true;


                            cv::imshow(window_name_hoop, std::get<0>(Hoop));
                            cv::moveWindow(window_name_hoop, 900, 700);
                        }
                    } else {
                        hoop_detected_count = 0;
                        this->Set_img_proc_hoop_det_(false);
                    }
                    Set_hoopcounter_det_flg_(hoop_detected_count);
                }
                break;

            case Interest_Object::Huddle:
                if (frameCounter % 3 == 0 || Get_UD_NeckAngle_() == HUDDLE_NECK) {
                // if (0) {
                    cv::Mat Roi_huddle = ROI_Rectangle(colorMat, 0, realsense_height, realsense_width / 2 - 100, realsense_width / 2 + 100);
            
                    // 불필요한 복사 제거
                    auto hsv_frame_yellow = extract_color(Roi_huddle, lower_bound_yellow, upper_bound_yellow);
                    auto hsv_frame_green = extract_color(colorMat, lower_bound_blue, upper_bound_blue);
            
                    // Detection
                    auto thresh_frame_yellow = detect_Huddle_areas(std::get<0>(hsv_frame_yellow), colorMat, yellow_color, threshold_value_yellow);
                    auto thresh_frame_green = detect_green_point(std::get<0>(hsv_frame_green), threshold_value_green);
                        
                    int YellowColorDetected = std::get<7>(thresh_frame_yellow);

                    if (YellowColorDetected > HUDDLE_AREA) {
                        Yellow_count++;
            
                        if (Yellow_count > 0) {
                            
                            noline_count = 0;
                            this->Set_img_proc_huddle_det_2d(true);
        
                            double gradient = std::get<1>(thresh_frame_yellow);
                            this->Set_gradient(gradient);
            
                            double huddle_angle_ = std::get<4>(thresh_frame_yellow);
                            Set_huddle_angle(huddle_angle_);

            
                            cv::Point huddle_center = std::get<5>(thresh_frame_yellow);
                            cv::circle(std::get<0>(thresh_frame_yellow), huddle_center, 2, CV_RGB(0, 255, 255), -1);
            
                            cv::Point foot_top_point = thresh_frame_green; 
                            cv::Point huddle_bottom_point = std::get<3>(thresh_frame_yellow);
                            cv::circle(std::get<0>(thresh_frame_yellow), huddle_bottom_point, 5, cv::Scalar(0, 0, 255), -1);
            
                            int minDistance = std::numeric_limits<int>::max();
                            cv::Point2f closestPoint;
            
                            for (const auto &point : std::get<6>(thresh_frame_yellow)) {
                                int distance = static_cast<int>(cv::norm(foot_top_point - point));
                                if (distance < minDistance) {
                                    minDistance = distance;
                                    closestPoint = point;
                                }
                            }
            
                            int _foot_huddle_distance = std::abs(foot_top_point.y - huddle_bottom_point.y);
                            Set_foot_huddle_distance(_foot_huddle_distance);
            
                            if (_foot_huddle_distance < HUDDLE_Y_MARGIN)
                            {
                                Set_contain_huddle_to_foot(true);
                            }
                            else
                            {
                                Set_contain_huddle_to_foot(false);
                            }

                            cv::line(std::get<0>(thresh_frame_yellow), foot_top_point, closestPoint, cv::Scalar(0, 0, 255), 3);
                            cv::putText(std::get<0>(thresh_frame_yellow), "Distance: " + std::to_string(minDistance), 
                            cv::Point(10, 175), cv::FONT_HERSHEY_SIMPLEX, 0.7, yellow_color, 2);
                            cv::putText(std::get<0>(thresh_frame_yellow), "Angle: " + std::to_string(huddle_angle_) + "deg", 
                            cv::Point(10, 200), cv::FONT_HERSHEY_SIMPLEX, 0.7, yellow_color, 2);
            
                            cv::imshow("hsv Frame_yellow", std::get<0>(thresh_frame_yellow));
                            cv::moveWindow("hsv Frame_yellow", 700, 0);
                        }
                    } else {
                        this->Set_img_proc_huddle_det_2d(false);
                        Yellow_count = 0; 
                
                    }
                }
                break;
        }

        // Line detection if no object detected
        if (!object_detected && (Get_UD_NeckAngle_() == 84)) {
        // if (1) {
            Line_frame = colorMat.clone();
            auto Roi_Line = ROI_Line(Line_frame, colorMat);
            auto hsv_frame_white = extract_color(std::get<0>(Roi_Line), lower_bound_white, upper_bound_white);
            auto thresh_frame_white = detect_Line_areas(std::get<0>(hsv_frame_white), colorMat, white_color, threshold_value_white);
            int WhiteColorDetected = std::get<2>(thresh_frame_white);

            auto hsv_frame_hsv = extract_color(std::get<0>(Roi_Line), {lowerH, lowerS, lowerV}, {upperH, upperS, upperV});
            cv::imshow(window_name_hsv, std::get<0>(hsv_frame_hsv));

            if (WhiteColorDetected > LINE_AREA) {
                White_count++;
                if (White_count > 30) {
                    noline_count = 0;
                    this->Set_img_proc_line_det(true);
                    this->Set_img_proc_no_line_det(false);
                    this->Set_gradient(std::get<1>(thresh_frame_white));
                    Yellow_count = 0;

                
                    cv::imshow("hsv Frame_white", std::get<0>(thresh_frame_white));
                    cv::moveWindow("hsv Frame_white", 0, 0);
                }
            }
            else if(WhiteColorDetected < LINE_AREA)
            {
                noline_count++;
                if(noline_count > 15)
                {
                    double gradient = std::get<1>(thresh_frame_white);
                    double tmp_delta_x = std::get<3>(thresh_frame_white);
                    
                    this->Set_img_proc_no_line_det(true);
                    this->Set_img_proc_line_det(false);
                    if (this->Get_img_proc_line_det() == false)
                    {
                        this->Set_gradient(gradient);
                        this->Set_delta_x(tmp_delta_x);
                    }
                }
            }
            
        }
    }
}
    
// ********************************************** GETTERS ************************************************** //

bool Img_proc::Get_img_proc_line_det() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_line_det_);
    return img_proc_line_det_;
}

bool Img_proc::Get_img_proc_no_line_det() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_no_line_det_);
    return img_proc_no_line_det_;
}

bool Img_proc::Get_img_proc_huddle_det_2d() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_huddle_det_2d);    
    return img_proc_huddle_det_2d_;
}

bool Img_proc::Get_img_proc_stop_det() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_stop_det_);
    return img_proc_stop_det_;
}

double Img_proc::Get_gradient() const
{
    std::lock_guard<std::mutex> lock(mtx_gradient);
    return gradient_;
}

double Img_proc::Get_delta_x() const
{
    std::lock_guard<std::mutex> lock(mtx_delta_x);
    return delta_x_;
}


double Img_proc::Get_huddle_distance() const
{
    std::lock_guard<std::mutex> lock(mtx_huddle_distance_);
    return huddle_distance_;
}

bool Img_proc::Get_contain_huddle_to_foot() const
{
    std::lock_guard<std::mutex> lock(mtx_contain_huddle_to_foot);
    return contain_huddle_to_foot_;
}

int Img_proc::Get_foot_huddle_distance() const
{
    std::lock_guard<std::mutex> lock(mtx_foot_huddle_distance_);
    return foot_huddle_distance_;
}

double Img_proc::Get_huddle_angle() const
{
    std::lock_guard<std::mutex> lock(mtx_huddle_angle_);
    return huddle_angle_;
}

bool Img_proc::Get_img_proc_Far_Hoop_det() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_far_hoop_det_);
    return img_proc_far_hoop_det_;
}

bool Img_proc::Get_img_proc_Adjust_det() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_adjust_det_);
    return img_proc_adjust_det_;
}

bool Img_proc::Get_img_proc_Shoot_det() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_shoot_det_);
    return img_proc_shoot_det_;
}

bool Img_proc::Get_img_proc_No_Hoop_det() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_no_hoop_det_);
    return img_proc_no_hoop_det_;
}

double Img_proc::Get_distance() const
{
    std::lock_guard<std::mutex> lock(mtx_distance);
    return distance_;
}

int8_t Img_proc::Get_img_proc_Adjust_number() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_adjust_number_);
    return img_proc_adjust_number_;
}

//double Img_proc::Get_gradient() const
//{
//    std::lock_guard<std::mutex> lock(mtx_gradient);
//    return gradient_;
//}

double Img_proc::Get_adjust_angle() const
{
    std::lock_guard<std::mutex> lock(mtx_adjust_angle);
    return adjust_angle_;
}

int Img_proc::Get_contain_adjust_to_foot() const
{
    std::lock_guard<std::mutex> lock(mtx_contain_adjust_to_foot);
    return contain_adjust_to_foot_;
}

double Img_proc::Get_ball_x() const
{
    std::lock_guard<std::mutex> lock(mtx_ball_x);
    return ball_x_;
}

double Img_proc::Get_ball_y() const
{
    std::lock_guard<std::mutex> lock(mtx_ball_y);
    return ball_y_;
}

double Img_proc::Get_ball_z() const
{
    std::lock_guard<std::mutex> lock(mtx_ball_z);
    return ball_z_;
}

double Img_proc::Get_hoop_x() const
{
    std::lock_guard<std::mutex> lock(mtx_hoop_x);
    return hoop_x_;
}

double Img_proc::Get_hoop_y() const
{
    std::lock_guard<std::mutex> lock(mtx_hoop_y);
    return hoop_y_;
}

double Img_proc::Get_hoop_z() const
{
    std::lock_guard<std::mutex> lock(mtx_hoop_z);
    return hoop_z_;
}

bool Img_proc::Get_img_proc_ball_det_() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_ball_det_);
    return img_proc_ball_det_;
}
bool Img_proc::Get_img_proc_hoop_det_() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_hoop_det_);
    return img_proc_hoop_det_;
}
bool Img_proc::Get_img_proc_huddle_det_() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_huddle_det_);
    return img_proc_huddle_det_;
}

bool Img_proc::Get_img_proc_ball_done_() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_ball_done_);
    return img_proc_ball_done_;
}
bool Img_proc::Get_img_proc_hoop_done_() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_hoop_done_);
    return img_proc_hoop_done_;
}

bool Img_proc::Get_img_proc_huddle_done_() const
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_huddle_done_);
    return img_proc_huddle_done_;
}

int Img_proc::Get_hoopcounter_det_flg_() const
{
    std::lock_guard<std::mutex> lock(mtx_hoop_counter);
    return hoop_counter_;
}


int Img_proc::Get_UD_NeckAngle_() const
{
    std::lock_guard<std::mutex> lock(mtx_UD_NeckAngle);
    return UD_NeckAngle_;
}


// ********************************************** SETTERS ************************************************** //

void Img_proc::Set_img_proc_line_det(bool img_proc_line_det)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_line_det_);
    this->img_proc_line_det_ = img_proc_line_det;
}

void Img_proc::Set_img_proc_no_line_det(bool img_proc_no_line_det)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_no_line_det_);
    this->img_proc_no_line_det_ = img_proc_no_line_det;
}

void Img_proc::Set_img_proc_huddle_det_2d(bool img_proc_huddle_det_2d)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_huddle_det_2d);
    this->img_proc_huddle_det_2d_ = img_proc_huddle_det_2d;
}

void Img_proc::Set_img_proc_stop_det(bool img_proc_stop_det)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_stop_det_);
    this->img_proc_stop_det_ = img_proc_stop_det;
}

void Img_proc::Set_gradient(double gradient)
{
    std::lock_guard<std::mutex> lock(mtx_gradient);
    this->gradient_ = gradient;
}

void Img_proc::Set_delta_x(double delta_x)
{
    std::lock_guard<std::mutex> lock(mtx_delta_x);
    this->delta_x_ = delta_x;
}

void Img_proc::Set_huddle_distance(double huddle_distance)
{
    std::lock_guard<std::mutex> lock(mtx_huddle_distance_);
    this->huddle_distance_ = huddle_distance;
}

void Img_proc::Set_contain_huddle_to_foot(bool contain_huddle_to_foot)
{
    std::lock_guard<std::mutex> lock(mtx_contain_huddle_to_foot);
    this->contain_huddle_to_foot_ = contain_huddle_to_foot;
}

void Img_proc::Set_foot_huddle_distance(int foot_huddle_distance)
{
    std::lock_guard<std::mutex> lock(mtx_foot_huddle_distance_);
    this->foot_huddle_distance_ = foot_huddle_distance;
}

void Img_proc::Set_huddle_angle(double huddle_angle)
{
    std::lock_guard<std::mutex> lock(mtx_huddle_angle_);
    this->huddle_angle_ = huddle_angle;
}

void Img_proc::Set_img_proc_Far_Hoop_det(bool img_proc_far_hoop_det)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_far_hoop_det_);
    this->img_proc_far_hoop_det_ = img_proc_far_hoop_det;
}

void Img_proc::Set_img_proc_Adjust_det(bool img_proc_adjust_det)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_adjust_det_);
    this->img_proc_adjust_det_ = img_proc_adjust_det;
}

void Img_proc::Set_img_proc_Shoot_det(bool img_proc_shoot_det)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_shoot_det_);
    this->img_proc_shoot_det_ = img_proc_shoot_det;
}

void Img_proc::Set_img_proc_No_Hoop_det(bool img_proc_no_hoop_det)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_no_hoop_det_);
    this->img_proc_no_hoop_det_ = img_proc_no_hoop_det;
}

//void Img_proc::Set_img_proc_stop_det(bool img_proc_stop_det)
//{
//    std::lock_guard<std::mutex> lock(mtx_img_proc_stop_det_);
//    this->img_proc_stop_det_ = img_proc_stop_det;
//}

//void Img_proc::Set_delta_x(double delta_x)
//{
//    std::lock_guard<std::mutex> lock(mtx_delta_x);
//    this->delta_x_ = delta_x;
//}

void Img_proc::Set_distance(double distance)
{
    std::lock_guard<std::mutex> lock(mtx_distance);
    this->distance_ = distance;
}

void Img_proc::Set_img_proc_adjust_number(int8_t img_proc_adjust_number)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_adjust_number_);
    this->img_proc_adjust_number_ = img_proc_adjust_number;
}

//void Img_proc::Set_gradient(double gradient)
//{
//    std::lock_guard<std::mutex> lock(mtx_gradient);
//    this->gradient_ = gradient;
//}

void Img_proc::Set_adjust_angle(double adjust_angle)
{
    std::lock_guard<std::mutex> lock(mtx_adjust_angle);
    this->adjust_angle_ = adjust_angle;
}

void Img_proc::Set_contain_adjust_to_foot(int contain_adjust_to_foot)
{
    std::lock_guard<std::mutex> lock(mtx_contain_adjust_to_foot);
    this->contain_adjust_to_foot_ = contain_adjust_to_foot;
}

void Img_proc::Set_ball_x(double ball_x)
{
    std::lock_guard<std::mutex> lock(mtx_ball_x);
    this->ball_x_ = ball_x;
}

void Img_proc::Set_ball_y(double ball_y)
{
    std::lock_guard<std::mutex> lock(mtx_ball_y);
    this->ball_y_ = ball_y;
}

void Img_proc::Set_ball_z(double ball_z)
{
    std::lock_guard<std::mutex> lock(mtx_ball_z);
    this->ball_z_ = ball_z;
}

void Img_proc::Set_img_proc_ball_det_(bool img_proc_ball_det_)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_ball_det_);
    this->img_proc_ball_det_ = img_proc_ball_det_;
}

void Img_proc::Set_img_proc_hoop_det_(bool img_proc_hoop_det_)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_hoop_det_);
    this->img_proc_hoop_det_ = img_proc_hoop_det_;
}

void Img_proc::Set_img_proc_huddle_det_(bool img_proc_huddle_det_)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_huddle_det_);
    this->img_proc_huddle_det_ = img_proc_huddle_det_;
}


void Img_proc::Set_img_proc_ball_done_(bool img_proc_ball_done_)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_ball_done_);
    this->img_proc_ball_done_ = img_proc_ball_done_;
}

void Img_proc::Set_img_proc_hoop_done_(bool img_proc_hoop_done_)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_hoop_done_);
    this->img_proc_hoop_done_ = img_proc_hoop_done_;
}
void Img_proc::Set_img_proc_huddle_done_(bool img_proc_huddle_done_)
{
    std::lock_guard<std::mutex> lock(mtx_img_proc_huddle_done_);
    this->img_proc_huddle_done_ = img_proc_huddle_done_;
}

void Img_proc::Set_hoop_x(double hoop_x)
{
    std::lock_guard<std::mutex> lock(mtx_hoop_x);
    this->hoop_x_ = hoop_x;
}

void Img_proc::Set_hoop_y(double hoop_y)
{
    std::lock_guard<std::mutex> lock(mtx_hoop_y);
    this->hoop_y_ = hoop_y;
}

void Img_proc::Set_hoop_z(double hoop_z)
{
    std::lock_guard<std::mutex> lock(mtx_hoop_z);
    this->hoop_z_ = hoop_z;
}

void Img_proc::Set_hoopcounter_det_flg_(int hoop_counter)
{
    std::lock_guard<std::mutex> lock(mtx_hoop_counter);
    this->hoop_counter_= hoop_counter;
}

void Img_proc::Set_UD_NeckAngle_(int UD_NeckAngle)
{
    std::lock_guard<std::mutex> lock(mtx_UD_NeckAngle);
    this->UD_NeckAngle_= UD_NeckAngle;
}
